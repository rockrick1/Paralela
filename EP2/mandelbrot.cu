#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <complex>
#include "png++/png.hpp"

using namespace std;

template<class real> // para trocar entre float e double
void mandelbrot_seq(char *argv[]){
	real c0r = stod(argv[1]);
	real c0i = stod(argv[2]);
	real c1r = stod(argv[3]);
	real c1i = stod(argv[4]);

	int W = stoi(argv[5]);
	int H = stoi(argv[6]);

	string CPU_GPU = argv[7];

	int threads = stoi(argv[8]);

	string saida = argv[9];

	real real_step = (c1r - c0r)/W;
	real imag_step = (c1i - c0i)/H;

	png::image< png::rgb_pixel > imagem(W, H);

	for (png::uint_32 y = 0; y < imagem.get_height(); ++y){
		for (png::uint_32 x = 0; x < imagem.get_width(); ++x){
			complex<real> point ( c0r+x*real_step , c0i+y*imag_step);
			const int M = 1000;

			// valor Zj que falhou
			// -1 se não tiver falhado
			int j = -1;

			//Valor da iteração passada
			complex<real> old_num (0,0);

			//Calcula o mandebrot
			for(int i = 1; i <= M; i++){

				old_num = old_num*old_num + point;

				if( (abs(old_num) > 2 )){
					j = i;
					break;
				}
			}

			if (j == -1){
				imagem.set_pixel(x, y, png::rgb_pixel(0, 0, 0));
			}
			else{
				png::uint_32 r = (M-j*255)/M;
				png::uint_32 g = (M-j*239)/M + 16;
				png::uint_32 b = (M-j*191)/M + 64;
				imagem.set_pixel(x, y, png::rgb_pixel(r, g, b));
			}
		}
	}

	imagem.write(saida);
}

template<class real> // para trocar entre float e double
void mandelbrot_omp(char *argv[]){
	real c0r = stod(argv[1]);
	real c0i = stod(argv[2]);
	real c1r = stod(argv[3]);
	real c1i = stod(argv[4]);

	int W = stoi(argv[5]);
	int H = stoi(argv[6]);

	string CPU_GPU = argv[7];

	int threads = stoi(argv[8]);

	string saida = argv[9];

	real real_step = (c1r - c0r)/W;
	real imag_step = (c1i - c0i)/H;

	png::image< png::rgb_pixel > imagem(W, H);
	png::uint_32 y;
	png::uint_32 x;

	#pragma omp parallel for collapse(2) num_threads(threads)
		for (y = 0; y < imagem.get_height(); ++y){
			for (x = 0; x < imagem.get_width(); ++x){
				complex<real> point ( c0r+x*real_step , c0i+y*imag_step);
				const int M = 1000;

				// valor Zj que falhou
				// -1 se não tiver falhado
				int j = -1;

				//Valor da iteração passada
				complex<real> old_num (0,0);

				//Calcula o mandebrot
				for(int i = 1; i <= M; i++){

					old_num = old_num*old_num + point;

					if( (abs(old_num) > 2 )){
						j = i;
						break;
					}
				}

				if (j == -1){
					imagem.set_pixel(x, y, png::rgb_pixel(0, 0, 0));
				}
				else{
					png::uint_32 r = (M-j*255)/M;
					png::uint_32 g = (M-j*239)/M + 16;
					png::uint_32 b = (M-j*191)/M + 64;
					imagem.set_pixel(x, y, png::rgb_pixel(r, g, b));
				}
			}
		}

	imagem.write(saida);
}

//Função que furtei do add.cu
void cudaAssert(hipError_t err)
{
    if (err != hipSuccess)
    {
        printf("Erro!\n");
        abort();
    }
}

__global__
template<class real> // para trocar entre float e double
void gpu_calculation(real c0r, real c0i, real real_step, real imag_step, real *results, unsigned n){

	// index = m*x + y
	const int globalIndex = blockDim.x*blockIdx.x + threadIdx.x;

	if (globalIndex < n)
        //calculo louco, deve dar varios errados, complexo nem deve existir em cuda
        complex<real> point ( c0r+blockIdx.x*real_step , c0i+threadsIdx.x*imag_step);
    	const int M = 1000;

		// valor Zj que falhou
		// -1 se não tiver falhado
		int j = -1;

		//Valor da iteração passada
		complex<real> old_num (0,0);

		//Calcula o mandebrot
		for(int i = 1; i <= M; i++){

			old_num = old_num*old_num + point;

			if( (abs(old_num) > 2 )){
				j = i;
				break;
			}
		}

		results[globalIndex] = j;

}

template<class real> // para trocar entre float e double
void mandelbrot_gpu(char *argv[]){
	real c0r = stod(argv[1]);
	real c0i = stod(argv[2]);
	real c1r = stod(argv[3]);
	real c1i = stod(argv[4]);

	int W = stoi(argv[5]);
	int H = stoi(argv[6]);

	string CPU_GPU = argv[7];

	int threads = stoi(argv[8]);

	string saida = argv[9];

	real real_step = (c1r - c0r)/W;
	real imag_step = (c1i - c0i)/H;

	png::image< png::rgb_pixel > imagem(W, H);
	png::uint_32 y;
	png::uint_32 x;

	//Cuda Stuff
	const int THREADS_PER_BLOCK = 128;
	const int NUM_BLOCKS = (W*H + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
	//Guarda os resultados calculados na GPU
	real *results = new real[W*H];

	//Coisas da memoria do cuda
	//Descomentar caso de erro <<<<<<<<<
	/*
	real cu_c0r;
	real cu_c0i;
	real cu_real_step;
	real cu_imag_step;
	*/
	real *cuda_results;

	//Aloca tudo
	/*
	cudaAssert(hipMalloc(&cu_c0r, sizeof(real)));
	cudaAssert(hipMalloc(&cu_c0i, sizeof(real)));
	cudaAssert(hipMalloc(&cu_real_step, sizeof(real)));
	cudaAssert(hipMalloc(&cu_imag_step, sizeof(real)));
	*/
	cudaAssert(hipMalloc(&cuda_results, W*H*sizeof(real)));

	//Copia tudo
	/*
	cudaAssert(hipMemcpy(cu_c0r, c0r, sizeof(real), hipMemcpyHostToDevice));
	cudaAssert(hipMemcpy(cu_c0i, c0i, sizeof(real), hipMemcpyHostToDevice));
	cudaAssert(hipMemcpy(cu_real_step, c1r, sizeof(real), hipMemcpyHostToDevice));
	cudaAssert(hipMemcpy(cu_imag_step, c1i, sizeof(real), hipMemcpyHostToDevice));
	*/

	//Dois problemas
	//1: não sei se templates funcionam
	//2: não sei se mandar diretamente c0r/etc funciona
	gpu_calculation<real><<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(c0r, c0i, real_step, imag_step, cuda_results, W*H);

	//Pega os resultados do Cuda e desaloca
	cudaAssert(hipMemcpy(results, cuda_results, W*H*sizeof(real), hipMemcpyDeviceToHost));
	/*
	hipFree(cu_c0r);
	hipFree(cu_c0i);
	hipFree(cu_real_step);
	hipFree(cu_imag_step);
	*/
	hipFree(cuda_results);

	const int N = W*H;
	int j; //para ficar parecido aos outros

	for(int p = 0; p < N; p++){
		//Acho que o X e o Y é algo assim
		int x = p/N;
		int y = p%N;

		j = results[p];

		if (j == -1)
			imagem.set_pixel(x, y, png::rgb_pixel(0, 0, 0));
		else{
			png::uint_32 r = (M-j*255)/M;
			png::uint_32 g = (M-j*239)/M + 16;
			png::uint_32 b = (M-j*191)/M + 64;
			imagem.set_pixel(x, y, png::rgb_pixel(r, g, b));
		}

	}

	imagem.write(saida);

	//Sempre bom desalocar
	delete [] results;
}

int main(int argc, char *argv[]){
	//processar os args
	//mbrot <C0_REAL> <C0_IMAG> <C1_REAL> <C1_IMAG> <W> <H> <CPU/GPU> <THREADS> <SAIDA>

	if(argc < 10){
		cout << "Incorrect Number of Args" << endl;
		cout << "Usage:" << endl;
		cout << "mbrot <C0_REAL> <C0_IMAG> <C1_REAL> <C1_IMAG> <W> <H> <CPU/GPU> <THREADS> <SAIDA>" << endl;
		return 0;
	}

	//mandelbrot_seq<float>(argv);
	//mandelbrot_omp<float>(argv);
	mandelbrot_gpu<float>(argv);
	return 0;

}